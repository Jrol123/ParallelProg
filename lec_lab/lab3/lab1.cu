#include<iostream>
#include<hip/hip_runtime.h>

__global__ void kernel ( void ) 
{
  int ID  = blockIdx.x * blockDim.x + threadIdx.x;
 // blockIdx.x номер блока
  //blockDim.x количество потоков в блоке
  //threadIdx.x номер потока в блоке
  
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
    printf("identificator, or number of thread is %d\n",ID); 
}

using namespace std;

int main(){
    cout << "Hello from CPU!" << endl;
  
    kernel <<< 1, 10 >>>(); //gpu 

    hipDeviceSynchronize();
    return 0;
}
