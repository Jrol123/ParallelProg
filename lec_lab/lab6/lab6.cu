#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

const int N = 4;
const int KERNEL = 2;
/**
 * @brief Matrix multiplication
 *
 * @param a left matrix
 * @param b right matrix
 * @param n matrix size
 * @param c result matrix
 *
 * @return __global__
 */
__global__ void matMul(float *a, float *b, int n, float *c)
{

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float sum = 0.0;

    int ia = n * (KERNEL * by + ty);
    int ib = KERNEL * bx + tx;

    for (int k = 0; k < n; ++k)
    {
        sum += a[ia + k] * b[k * n + ib];
    }

    int ic = n * (KERNEL * by + ty) + (KERNEL * bx + tx);
    c[ic] = sum;
}

int main()
{
    // Создание матриц
    float(*A)[N] = new float[N][N];
    float(*B)[N] = new float[N][N];
    float(*C)[N] = new float[N][N];

    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            A[i][j] = (i + j) * 1.0;
            B[i][j] = (i + j) * 1.0;
            cout << (i + j) * 1.0 << " ";
        }
        cout << "\n";
    }

    // Создание матриц на видеокарте
    float *dev_A, *dev_B, *dev_C;
    hipMalloc((void **)&dev_A, N * N * sizeof(float));
    hipMalloc((void **)&dev_B, N * N * sizeof(float));
    hipMalloc((void **)&dev_C, N * N * sizeof(float));

    hipMemcpy(dev_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Вычисление

    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    hipEventRecord(startEvent, 0);

    dim3 threads(KERNEL, KERNEL);
    dim3 grid((N + KERNEL - 1) / KERNEL, (N + KERNEL - 1) / KERNEL);

    matMul<<<grid, threads>>>(dev_A, dev_B, N, dev_C);

    hipMemcpy(C, dev_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    // Вывод

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);

    cout << "Время выполнения: " << elapsedTime << " миллисекунд" << endl;

    cout << "Результат матричного умножения C:" << endl;
    for (int i = 0; i < N; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            cout << C[i][j] << ' ';
        }
        cout << endl;
    }

    // Конец

    delete[] A;
    delete[] B;
    delete[] C;
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    return 0;
}
